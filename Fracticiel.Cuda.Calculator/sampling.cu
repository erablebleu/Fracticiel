#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include "include.h"
#include "common.h"

__global__ void kernel_multisampling(uint32_t* result, uint32_t* data, int32_t dW, int32_t dH, int32_t sampling) {
   int x = (blockIdx.x * blockDim.x + threadIdx.x);
   int y = (blockIdx.y * blockDim.y + threadIdx.y);
   uint64_t val = 0;

   if (x >= dW || y >= dH)
      return;

   for (int i = 0; i < sampling; i++)
      for (int j = 0; j < sampling; j++) 
         val += data[x * sampling + i + (y * sampling + j) * dW * sampling];

   result[x + y * dW] = val / sampling;
}

int32_t multisampling(uint32_t* result, uint32_t* data, int32_t dW, int32_t dH, int32_t multisampling) {
   uint32_t* cuda_result = 0;
   uint32_t* cuda_data = 0;

   CUDA_ASSERT_SUCCESS(hipSetDevice(0));
   CUDA_ASSERT_SUCCESS(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
   CUDA_ASSERT_SUCCESS(hipMalloc((void**)&cuda_result, dW * dH * sizeof(uint32_t)));
   CUDA_ASSERT_SUCCESS(hipMalloc((void**)&cuda_data, dW * dH * multisampling * multisampling * sizeof(uint32_t)));
   CUDA_ASSERT_SUCCESS(hipMemcpy(cuda_data, data, dW * multisampling * dH * multisampling * sizeof(uint32_t), hipMemcpyHostToDevice));
   dim3 threadsPerBlock(16, 16);
   dim3 numBlocks(dW / threadsPerBlock.x + 1, dH / threadsPerBlock.y + 1);
   kernel_multisampling << <numBlocks, threadsPerBlock >> > (cuda_result, cuda_data, dW, dH, multisampling);
   CUDA_ASSERT_SUCCESS(hipGetLastError());
   CUDA_ASSERT_SUCCESS(hipDeviceSynchronize());
   CUDA_ASSERT_SUCCESS(hipMemcpy(result, cuda_result, dW * dH * sizeof(uint32_t), hipMemcpyDeviceToHost));
   CUDA_ASSERT_SUCCESS(hipFree(cuda_result));
   CUDA_ASSERT_SUCCESS(hipDeviceReset());

   return hipSuccess;
}