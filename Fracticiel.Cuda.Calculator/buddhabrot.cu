#include "hip/hip_runtime.h"

#include <stdio.h>

#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include ""
#include "include.h"
#include "common.h"

__global__ void kernel_setup_curand(hiprandState* state) {
   int idx = (blockIdx.x * blockDim.x + threadIdx.x);
   hiprand_init(idx, idx, 0, &state[idx]);
}

QUALIFIERS bool kernel_buddhabrot_isDivergent(double cx, double cy, int32_t divergenceCount, double divergenceMagnitude) {
   double x = 0.0;
   double y = 0.0;
   double tmp;
   double magnitude;

   for (int k = 0; k < divergenceCount; k++)
   {
      tmp = x * x - y * y + cx;
      y = 2.0f * x * y + cy;
      x = tmp;
      magnitude = x * x + y * y;

      if (magnitude > divergenceMagnitude)
         return true;
   }
   return false;
}

QUALIFIERS void kernel_buddhabrot(int32_t* result,
                                  int32_t dW, int32_t dH, 
                                  double x0, double y0, 
                                  double cx, double cy,
                                  double res, 
                                  int32_t divergenceCount,
                                  double divergenceMagnitude) {
   int i, j;
   double x = 0.0;
   double y = 0.0;
   double tmp;
   double magnitude;

   if(!kernel_buddhabrot_isDivergent(cx, cy, divergenceCount, divergenceMagnitude))
      return;

   x = 0.0;
   y = 0.0;

   for (int k = 0; k < divergenceCount; k++)
   {
      tmp = x * x - y * y + cx;
      y = 2.0f * x * y + cy;
      x = tmp;
      magnitude = x * x + y * y;

      i = (x - x0) / res;
      j = (y - y0) / res;

      if (i >= 0 && i < dW
         && j >= 0 && j < dH) {
         result[j * dW + i]++;
      }

      if (magnitude > divergenceMagnitude)
         break;
   }
}

__global__ void kernel_buddhabrot(hiprandState* state,
                                  int32_t* result,
                                  int32_t dW, int32_t dH,
                                  double x0, double y0,
                                  double resolution,
                                  int32_t divergenceCount,
                                  double divergenceMagnitude,
                                  int32_t startPointCount) {
   int idx = (blockIdx.x * blockDim.x + threadIdx.x);

   for(int l = 0; l < startPointCount; l++) {
      kernel_buddhabrot(result, dW, dH, x0, y0, 
                        (double)(-2.0) + (double)4.0 * hiprand_uniform_double(&state[idx]),
                        (double)(-2.0) + (double)4.0 * hiprand_uniform_double(&state[idx]),
                        resolution,
                        divergenceCount,
                        divergenceMagnitude);
   }
}

int32_t buddhabrot(int32_t* result, const DataBlock* block, const Settings_Buddhabrot* settings) {
   int32_t* cuda_result = 0;
   hiprandState* d_state = 0;
   int32_t blockCount = 256;
   int32_t threadCount = 256;

   CUDA_ASSERT_SUCCESS(hipSetDevice(0));
   CUDA_ASSERT_SUCCESS(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
   CUDA_ASSERT_SUCCESS(hipMalloc(&d_state, blockCount * threadCount * sizeof(hiprandState)));
   CUDA_ASSERT_SUCCESS(hipMalloc((void**)&cuda_result, block->Width * block->Height * sizeof(int32_t)));
   CUDA_ASSERT_SUCCESS(hipDeviceSynchronize());
   kernel_setup_curand << <blockCount, threadCount >> > (d_state);
   kernel_buddhabrot << <blockCount, threadCount >> > (d_state,
                                        cuda_result, 
                                        block->Width, block->Height, 
                                        block->X, block->Y, 
                                        block->Resolution,
                                        settings->LoopCount, 
                                        settings->Magnitude,
      settings->StartPointCount);
   CUDA_ASSERT_SUCCESS(hipGetLastError());
   CUDA_ASSERT_SUCCESS(hipDeviceSynchronize());
   CUDA_ASSERT_SUCCESS(hipMemcpy(result, cuda_result, block->Width * block->Height * sizeof(int32_t), hipMemcpyDeviceToHost));
   CUDA_ASSERT_SUCCESS(hipFree(cuda_result));
   CUDA_ASSERT_SUCCESS(hipFree(d_state));
   CUDA_ASSERT_SUCCESS(hipDeviceReset());

   return hipSuccess;
}