#include "hip/hip_runtime.h"

#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include "include.h"
#include "common.h"

__global__ void kernel_mandelbrot(uint32_t* result, uint32_t dW, uint32_t dH, double x0, double y0, double res, uint32_t maxLoopCnt, double maxMagnitude) {
   int i = (blockIdx.x * blockDim.x + threadIdx.x);
   int j = (blockIdx.y * blockDim.y + threadIdx.y);

   if (i >= dW || j >= dH)
      return;

   double cx = x0 + (double)i * res;
   double cy = y0 + (double)j * res;
   uint32_t* p = &result[j * dW + i];
   double x = 0.0f;
   double y = 0.0f;
   double tmp;

   maxMagnitude *= maxMagnitude;

   while ((x * x + y * y) <= maxMagnitude && *p < maxLoopCnt)
   {
      tmp = x * x - y * y + cx;
      y = 2.0f * x * y + cy;
      x = tmp;
      (*p)++;
   }
}

int32_t mandelbrot(uint32_t* result, uint32_t dW, uint32_t dH, double x, double y, double res, uint32_t maxLoopCnt, double maxMagnitude) {
   uint32_t* cuda_result = 0;

   CUDA_ASSERT_SUCCESS(hipSetDevice(0));
   CUDA_ASSERT_SUCCESS(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
   CUDA_ASSERT_SUCCESS(hipMalloc((void**)&cuda_result, dW * dH * sizeof(uint32_t)));
   dim3 threadsPerBlock(16, 16);
   dim3 numBlocks(dW / threadsPerBlock.x + 1, dH / threadsPerBlock.y + 1);
   kernel_mandelbrot << <numBlocks, threadsPerBlock >> > (cuda_result, dW, dH, x, y, res, maxLoopCnt, maxMagnitude);
   CUDA_ASSERT_SUCCESS(hipGetLastError());
   CUDA_ASSERT_SUCCESS(hipDeviceSynchronize());
   CUDA_ASSERT_SUCCESS(hipMemcpy(result, cuda_result, dW * dH * sizeof(uint32_t), hipMemcpyDeviceToHost));
   CUDA_ASSERT_SUCCESS(hipFree(cuda_result));
   CUDA_ASSERT_SUCCESS(hipDeviceReset());

   return hipSuccess;
}